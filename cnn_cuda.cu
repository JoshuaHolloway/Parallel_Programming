#include "hip/hip_runtime.h"
#include "header.h"
//===============
struct FeatureMap
{
	float* val = nullptr;
	size_t rows;
	size_t cols;
	size_t channels;
	size_t num_elems;

	FeatureMap(size_t rows, size_t cols, size_t channels)
	{
		num_elems = rows * cols * channels;
		this->rows = rows;
		this->cols = cols;
		this->channels = channels;
		val = new float[num_elems];
	}


	// Easier indexing is achieved with (channel, row, col)
	void set(size_t i, size_t j, size_t k, float val)
	{
		this->val[i * rows * cols + j * cols + k] = val;
	}

	float at(size_t i, size_t j, size_t k) //(channel, row, col)
	{
		return val[i * rows * cols + j * cols + k];
	}

	void ones()
	{
		for (int i = 0; i < channels; ++i)
		{
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					this->val[i * rows * cols + j * cols + k] = 1;
				}
			}
		}
	}

	void zeros()
	{
		for (int i = 0; i < channels; ++i)
		{
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					this->val[i * rows * cols + j * cols + k] = 0;
				}
			}
		}
	}

	void count()
	{
		for (int i = 0; i < channels; ++i)
		{
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					this->val[i * rows * cols + j * cols + k] = i * rows * cols + j * cols + k;
				}
			}
		}
	}

	void print()
	{
		for (int i = 0; i < channels; ++i)
		{
			cout << "\n ----------------- \n";
			cout << "Slice " << i << " \n";
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					cout << val[i * rows * cols + j * cols + k] << " ";
				}
				cout << "\n";
			}
		}
	}
};
//---------------
struct Tensor
{

	size_t dim1;
	size_t dim2;
	size_t dim3;
	size_t dim4;

	// pixel data
	float* val = nullptr;

	// (filters, channels, rows, cols)
	size_t filters;		// dim 1
	size_t channels;	// dim 2
	size_t rows;			// dim 3
	size_t cols;			// dim 4

							// total number of pixels
	size_t num_elems;


	// Vector of feature maps - tensor = set of feature maps:
	std::vector<FeatureMap> tensor;

	// Constructor and destructor
	Tensor(size_t filters, size_t channels, size_t rows, size_t cols)
	{
		num_elems = rows * cols * channels * filters;

		val = new float[num_elems];

		this->filters = filters;			this->dim1 = filters;
		this->channels = channels;		this->dim2 = channels;
		this->rows = rows;						this->dim3 = rows;
		this->cols = cols;						this->dim4 = cols;
	}
	~Tensor()
	{
		delete[] val;
	}

	// Easier indexing is achieved with (output_channel, input_channel, row, col)
	void set(size_t i, size_t j, size_t k, size_t l, float val)
	{
		// i: dim1 - filters
		// j: dim2 - channels
		// k: dim3 - rows
		// l: dim4 - cols

		// 3D linear index: i * rows * cols + j * cols + k
		// i * cols * rows + j * cols + k
		// (i * dim3 * dim2) + (j * dim3) + k

		// 4D linear index:
		// (i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l

		this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] = val;
	}

	float at(size_t i, size_t j, size_t k, size_t l) //(filters, channel, row, col)
	{
		return val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l];
	}

	void ones()
	{
		for (int i = 0; i < dim1; ++i)
		{
			for (int j = 0; j < dim2; ++j)
			{
				for (int k = 0; k < dim3; ++k)
				{
					for (int l = 0; l < dim4; ++l)
						this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] = 1;
				}
			}
		}
	}

	void zeros()
	{
		for (int i = 0; i < dim1; ++i)
		{
			for (int j = 0; j < dim2; ++j)
			{
				for (int k = 0; k < dim3; ++k)
				{
					for (int l = 0; l < dim4; ++l)
						this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] = 0;
				}
			}
		}
	}

	void count()
	{
		for (int i = 0; i < dim1; ++i)
		{
			for (int j = 0; j < dim2; ++j)
			{
				for (int k = 0; k < dim3; ++k)
				{
					for (int l = 0; l < dim4; ++l)
					{
						this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l]
							= (i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l;

					}
				}
			}
		}
	};

	void print()
	{
		for (int i = 0; i < filters; ++i)
		{
			cout << "\n ----------------- \n";
			cout << "Volume " << i << " \n";
			for (int j = 0; j < channels; ++j)
			{
				cout << "\n ----------------- \n";
				cout << "Slice " << i << " \n";
				for (int k = 0; k < rows; ++k)
				{
					for (int l = 0; l < cols; ++l)
					{
						cout << val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] << " ";
					}
					cout << "\n";
				}
			}
		}
	}
};
//-------------------------------------
FeatureMap conv(FeatureMap x, Tensor h)
{
	// 'same' 2D conv with 3D feature maps with implicit matrix slice addition
	// Zero-padding is also implicit
	//
	// Input: One 3D feature map and one 4D tensor (set of filters)
	// Output: One 3D feature map
	FeatureMap y(x.rows, x.cols, h.filters);
	for (int idq = 0; idq < h.filters; ++idq) // out_channels
	{
		for (int idy = 0; idy < x.rows; ++idy) // out_rows
		{
			for (int idx = 0; idx < x.cols; ++idx) // out_cols
			{
				float Pvalue = 0.0f;
				for (int idz = 0; idz < x.channels; ++idz) // input_channels
				{

					int M_start_point = idy - h.rows / 2;
					int N_start_point = idx - h.cols / 2;
					for (int i = 0; i < h.rows; ++i) // filter_rows
					{
						for (int j = 0; j < h.cols; ++j) // filter_cols
						{
							if ((M_start_point + i >= 0 && M_start_point + i < x.rows)
								&& (N_start_point + j >= 0 && N_start_point + j < x.cols))
							{
								Pvalue += x.at(idz, M_start_point + i, N_start_point + j) * h.at(idq, idz, i, j);
							}
						}
					}
					y.set(idq, idy, idx, Pvalue);
				}
			}
		}
	}
	return y;
}
//-------------------------------
FeatureMap pool_ave(FeatureMap x)
{
	const size_t H = x.rows;
	const size_t W = x.cols;
	const size_t M = x.channels;
	const size_t K = 2; // downsampling factor

	FeatureMap S(H / K, W / K, x.channels); // rows, cols, channels

	for (int m = 0; m < M; ++m)  // channels
	{
		for (int h = 0; h < H / K; ++h) // rows
		{
			for (int w = 0; w < W / K; ++w)
			{
				float temp = 0.0f;
				for (int p = 0; p < K; ++p)
				{
					for (int q = 0; q < K; ++q)
					{
						temp += x.at(m, K*h + p, K*w + q) / float(K*K);
					}
				}
				S.set(m, h, w, temp);
			}
		}
	}

	return S;
}
//-------------------------------
FeatureMap pool_max(FeatureMap x)
{
	// downsampling factor:
	const size_t K = 2;

	// output downsampled feature map:
	FeatureMap y(x.rows / K, x.cols / K, x.channels); // rows, cols, channels

	for (int i = 0; i < x.channels; ++i)
	{
		for (int j = 0; j < x.rows; j += K)
		{
			for (int k = 0; k < x.cols; k += K)
			{
				// Search inside the KxK block for max value
				int max = 0;
				for (int jj = j; jj < j + K; ++jj)
				{
					for (int kk = k; kk < k + K; ++kk)
					{
						if (jj == j && kk == k)
						{
							max = x.at(i, jj, kk);
						}
						else
						{
							if (x.at(i, jj, kk) > max)
								max = x.at(i, jj, kk);
						} // end if-else
					}// end for over kk
				} // end for over jj
				  //y[i][j / 2][k / 2] = max;
				y.set(i, j / 2, k / 2, max);
			} // end for over k
		} // end for over j
	} // end for over i
	return y;
}
//---------------------------
FeatureMap relu(FeatureMap z)
{
	for (int i = 0; i < z.channels; ++i)
	{
		for (int j = 0; j < z.rows; ++j)
		{
			for (int k = 0; k < z.cols; ++k)
			{
				// Leaky ReLu
				if (z.at(i, j, k) < 0)
					z.set(i, j, k, 0.1*z.at(i, j, k));
			}
		}
	}
	return z;
}
//=============================================================================
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
//-----------------------------------------------------------------------------
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
//--------
//--------------------------------------
void save_image(const char* output_filename,
	float* buffer,
	int height,
	int width) {
	cv::Mat output_image(height, width, CV_32FC3, buffer);
	// Make negative values zero.
	cv::threshold(output_image,
		output_image,
		/*threshold=*/0,
		/*maxval=*/0,
		cv::THRESH_TOZERO);
	cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
	output_image.convertTo(output_image, CV_8UC3);
	cv::imwrite(output_filename, output_image);
}
//--------------------------------------
int kernel_wrapper(const cv::Mat& image)
{
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

	// Create context object
	hipdnnHandle_t cudnn;
	checkCUDNN(hipdnnCreate(&cudnn));

	// Input tensor
	hipdnnTensorDescriptor_t input_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
		/*format=*/HIPDNN_TENSOR_NHWC,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/3,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// Output tensor
	hipdnnTensorDescriptor_t output_descriptor;
	checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
	checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
		/*format=*/HIPDNN_TENSOR_NHWC,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*batch_size=*/1,
		/*channels=*/3,
		/*image_height=*/image.rows,
		/*image_width=*/image.cols));

	// Filter tensor
	hipdnnFilterDescriptor_t kernel_descriptor;
	checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
	checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
		/*dataType=*/HIPDNN_DATA_FLOAT,
		/*format=*/HIPDNN_TENSOR_NCHW,
		/*out_channels=*/3,
		/*in_channels=*/3,
		/*kernel_height=*/3,
		/*kernel_width=*/3));

	// Describe the conv kernel
	hipdnnConvolutionDescriptor_t convolution_descriptor;
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
	checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
		/*pad_height=*/1,
		/*pad_width=*/1,
		/*vertical_stride=*/1,
		/*horizontal_stride=*/1,
		/*dilation_height=*/1,
		/*dilation_width=*/1,
		/*mode=*/HIPDNN_CROSS_CORRELATION,
		/*computeType=*/HIPDNN_DATA_FLOAT));

	// More detailed description of the convolution algorithm we want to use:
	hipdnnConvolutionFwdAlgo_t convolution_algorithm;
	checkCUDNN(
		hipdnnGetConvolutionForwardAlgorithm(cudnn,
			input_descriptor,
			kernel_descriptor,
			convolution_descriptor,
			output_descriptor,
			HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
			/*memoryLimitInBytes=*/0,
			&convolution_algorithm));

	// Physical memory to operate on
	size_t workspace_bytes = 0;
	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
		input_descriptor,
		kernel_descriptor,
		convolution_descriptor,
		output_descriptor,
		convolution_algorithm,
		&workspace_bytes));
	std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
		<< std::endl;

	// Dimensions from OpenCV input image
	int batch_size = 1;
	int channels = image.channels();
	int height = image.rows;
	int width = image.cols;
	cout << "\n batch_size = " << batch_size << " channels = " << channels << " rows = " << height << " cols = " << width << "\n";

	// Dimensions from cuDNN for confirmation
	hipdnnGetConvolution2dForwardOutputDim(
		convolution_descriptor, 
		input_descriptor,
		kernel_descriptor,
		&batch_size,
		&channels,
		&height,
		&width);
	cout << "\n batch_size = " << batch_size << " channels = " << channels << " rows = " << height << " cols = " << width << "\n";
	

	// Allocate device memory
	void* d_workspace{ nullptr };
	hipMalloc(&d_workspace, workspace_bytes);

	int image_bytes = batch_size * channels * height * width * sizeof(float);

	float* d_input{ nullptr };
	hipMalloc(&d_input, image_bytes);
	hipMemcpy(d_input, image.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);

	float* d_output{ nullptr };
	hipMalloc(&d_output, image_bytes);
	hipMemset(d_output, 0, image_bytes);

	// Mystery kernel
	const float kernel_template[3][3] = {
		{ 1,  1, 1 },
		{ 1, -8, 1 },
		{ 1,  1, 1 }
	};

	float h_kernel[3][3][3][3];
	for (int kernel = 0; kernel < 3; ++kernel) {
		for (int channel = 0; channel < 3; ++channel) {
			for (int row = 0; row < 3; ++row) {
				for (int column = 0; column < 3; ++column) {
					h_kernel[kernel][channel][row][column] = kernel_template[row][column];
				}
			}
		}
	}

	float* d_kernel{ nullptr };
	hipMalloc(&d_kernel, sizeof(h_kernel));
	hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

	// Do conv
	const float alpha = 1, beta = 0;
	checkCUDNN(hipdnnConvolutionForward(cudnn,
		&alpha,
		input_descriptor,
		d_input,
		kernel_descriptor,
		d_kernel,
		convolution_descriptor,
		convolution_algorithm,
		d_workspace,
		workspace_bytes,
		&beta,
		output_descriptor,
		d_output));

	// Copy DEVICE -> HOST
	float* h_output = new float[image_bytes];
	hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

	// Do something with h_output ...
	save_image("convolved_img.jpg",
		h_output,
		height,
		width);
	//void save_image(const char* output_filename,
	//	float* buffer,
	//	int height,
	//	int width);

	// Free memory
	delete[] h_output;
	hipFree(d_kernel);
	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_workspace);

	hipdnnDestroyTensorDescriptor(input_descriptor);
	hipdnnDestroyTensorDescriptor(output_descriptor);
	hipdnnDestroyFilterDescriptor(kernel_descriptor);
	hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

	hipdnnDestroy(cudnn);


	// Custom CNN
	const size_t R[26] = { 416, 416, 208, 208, 104, 104, 104, 104, 52, 52, 52, 52, 26, 26, 26, 26, 26, 26,13,13,13,13,13,13,13,13 }; // Rows    in each 2D matrix slice in each 3D feature map
	const size_t C[26] = { 416, 416, 208, 208, 104, 104, 104, 104, 52, 52, 52, 52, 26, 26, 26, 26, 26, 26,13,13,13,13,13,13,13,13 }; // Columns in each 2D matrix slice in each 3D feature map
	const size_t K = 3; // Filter size

	size_t D[25];
	D[0] = 3;
	D[1] = 32;
	D[2] = 32;
	D[3] = 64;
	D[4] = 64;
	D[5] = 128;
	D[6] = 64;
	D[7] = 128;
	D[8] = 128;
	D[9] = 256;
	D[10] = 128;
	D[11] = 256;
	D[12] = 256;
	D[13] = 512;
	D[14] = 256;
	D[15] = 512;
	D[16] = 256;
	D[17] = 512;
	D[18] = 512;
	D[19] = 1024;
	D[20] = 512;
	D[21] = 1024;
	D[22] = 512;
	D[23] = 1024;
	D[24] = 1024;
	D[25] = 1024;

	/// Section 1 - layers 1,2: conv-pool
	FeatureMap X(R[0], C[0], D[0]);     X.count();
	Tensor H1(D[1], D[0], K, K); /* */ H1.ones(); // Layer 1
	//Tensor H2(D[2], D[1], K, K); /* */ H2.ones(); // Layer 2 - Pool

	/// Section 2 - layers 3,4: conv-pool
	Tensor H3(D[3], D[2], K, K); /* */ H3.ones(); // Layer 3
	//Tensor H4(D[4], D[3], K, K); /* */ H4.ones(); // Layer 4 - Pool

/*
	/// Section 3 - layers 5-8: conv(x3)-pool
	Tensor H5(D[5], D[4], K, K);  H5.ones(); // Layer 5
	Tensor H6(D[6], D[5], K, K);  H6.ones(); // Layer 6
	Tensor H7(D[7], D[6], K, K);  H7.ones(); // Layer 7
	//Tensor H8(D[8], D[7], K, K);  H8.ones(); // Layer 8 - Pool

	/// Section 4 - layers 9-12: conv(x3)-pool
	Tensor H9(D[9], D[8], K, K);     H9.ones(); // Layer 9
	Tensor H10(D[10], D[9], K, K);   H10.ones(); // Layer 10
	Tensor H11(D[11], D[10], K, K);  H11.ones(); // Layer 11
	//Tensor H12(D[12], D[11], K, K);  H12.ones(); // Layer 12 - Pool

	/// Section 5 - layers 13-18: conv(5x)-pool
	Tensor H13(D[13], D[12], K, K);  H13.ones(); // Layer 13
	Tensor H14(D[14], D[13], K, K);  H14.ones(); // Layer 14
	Tensor H15(D[15], D[14], K, K);  H15.ones(); // Layer 15
	Tensor H16(D[16], D[15], K, K);  H16.ones(); // Layer 16
	Tensor H17(D[17], D[16], K, K);  H17.ones(); // Layer 17
	//Tensor H18(D[18], D[17], K, K);  H18.ones(); // Layer 18 - Pool

/// Section 6 - layers 19-23: conv(5x)
	Tensor H19(D[18], D[17], K, K);  H19.ones(); // Layer 19
	Tensor H20(D[19], D[18], K, K);  H20.ones(); // Layer 20
	Tensor H21(D[20], D[19], K, K);  H21.ones(); // Layer 21 
	Tensor H22(D[21], D[20], K, K);  H22.ones(); // Layer 22
	Tensor H23(D[22], D[21], K, K);  H23.ones(); // Layer 23
*/
	// Start CPU Timing
	LARGE_INTEGER start_CPU, end_CPU, frequency_CPU;
	double milliseconds_CPU, seconds_CPU, minutes_CPU;
	QueryPerformanceFrequency(&frequency_CPU);
	QueryPerformanceCounter(&start_CPU);

	// |-----------section 1-----------|--------section 2---------|------------------------section 3------------------------|------------------section 4------------------------|------------------------------section 5------------------------------------|
	// Layer:   1              2              3            4              5              6             7              8             9           10          11            12          13            14           15           16           17           18
	//         conv           max           conv          max           conv           conv          conv            max          conv         conv        conv          max         conv          conv         conv         conv         conv         max 
	// 416x416x3 -> 416x416x32 -> 208x208x32 -> 208x208x64 -> 104x104x64 -> 104x104x128 -> 104x104x64 ->  104x104x128 -> 52x52x128 -> 52x52x256 -> 52x52x128 -> 52x52x256 -> 26x26x256 -> 26x26x512 -> 26x26x256 -> 26x26x512 -> 26x26x256 -> 26x26x512 -> ...
	//  D[0]=3       D[1]=32        D[2]=32       D[3]=64       D[4]=64       D[5]=128       D[6]=64        D[7]=128     D[8]=128      D[9]=256    D[10]=128    D[11]=256    D[12]=256    D[13]=512    D[14]=256    D[15]=512    D[16]=256    D[17]=512
	
	//                                                         FEATURE-EXTRACTION   | DETECTION
	//                conv         conv          conv          conv         conv          conv          conv          route conv reorg route conv conv detection
	// ...-> 13x13x512 -> 13x13x1024 -> 13x13x512 -> 13x13x1024 -> 13x13x512 -> 13x13x1024 -> 13x13x1024 -> 13x13x1024 -> 
	//       D[18]=512    D[19]=1024    D[20]=512    D[21]=1024    D[22]=512    D[23]=1024    D[24]=1024    D[25]=1024
	//  |---------------------------section 6---------------------------------------|------------------------------section 5------------------------------------|

	// -----------
	// Section 1:
	// -----------
	cout << "Section 1: layers 1-2" << R[0] << "x" << C[0] << "x" << D[0] << " -> " << R[1] << "x" << C[1] << "x" << D[1] << " -> " << R[2] << "x" << C[2] << "x" << D[2] << "\n";
	cout << "From Darknet: 416x416x3 -> 416x416x32 -> 208x208x32 \n";
	FeatureMap A1 = pool_max(relu(conv(X, H1)));

	// -----------
	// Section 2:
	// -----------
	cout << "\nSection 2: layers 3-4" << R[2] << "x" << C[2] << "x" << D[2] << " -> " << R[3] << "x" << C[3] << "x" << D[3] << " -> " << R[4] << "x" << C[4] << "x" << D[4] << "\n";
	cout << "From Darknet: 208x208x32 -> 208x208x64 -> 104x104x64 \n";
	FeatureMap A3 = pool_max(relu(conv(A1, H3)));

	/*

	// -----------
	// Section 3:
	// -----------
	cout << "\nSection 3: layers 5-8" << R[4] << "x" << C[4] << "x" << D[4] << " -> " << R[5] << "x" << C[5] << "x" << D[5] << " -> " << R[6] << "x" << C[6] << "x" << D[6]
		<< " -> " << R[7] << "x" << C[7] << "x" << D[7] << " -> " << R[8] << "x" << C[8] << "x" << D[8] << "\n";
	cout << "From Darknet: 104x104x64 -> 104x104x128 -> 104x104x64 ->  104x104x128 -> 52x52x128 \n";
	FeatureMap A5 = relu(conv(A3, H5));
	FeatureMap A6 = relu(conv(A5, H6));
	FeatureMap A7 = relu(conv(A6, H7));
	FeatureMap A8 = pool_max(A7);

	// -----------
	// Section 4:
	// -----------
	cout << "\nSection 4: layers 9-12" << R[8] << "x" << C[8] << "x" << D[8] << " -> " << R[9] << "x" << C[9] << "x" << D[9] << " -> " << R[10] << "x" << C[10] << "x" << D[10]
		<< " -> " << R[11] << "x" << C[11] << "x" << D[11] << " -> " << R[12] << "x" << C[12] << "x" << D[12] << "\n";
	cout << "From Darknet: 52x52x128 -> 52x52x256 -> 52x52x128 -> 52x52x256 -> 26x26x256 \n";
	FeatureMap A9 = relu(conv(A8, H9));
	FeatureMap A10 = relu(conv(A9, H10));
	FeatureMap A11 = relu(conv(A10, H11));
	FeatureMap A12 = pool_max(A11);

	// -----------
	// Section 5:
	// -----------
	cout << "\nSection 5: layers 13-18" << R[12] << "x" << C[12] << "x" << D[12] << " -> " << R[13] << "x" << C[13] << "x" << D[13] << " -> " << R[14] << "x" << C[14] << "x" << D[14]
		<< " -> " << R[15] << "x" << C[15] << "x" << D[15] << " -> " << R[16] << "x" << C[16] << "x" << D[16]
		<< " -> " << R[17] << "x" << C[17] << "x" << D[17] << " -> " << R[18] << "x" << C[18] << "x" << D[18] << "\n";
	cout << "From Darknet: 26x26x256 -> 26x26x512 -> 26x26x256 -> 26x26x512 -> 26x26x256 -> 26x26x512 -> 13x13x512 \n";
	FeatureMap A13 = relu(conv(A12, H13));
	FeatureMap A14 = relu(conv(A13, H14));
	FeatureMap A15 = relu(conv(A14, H15));
	FeatureMap A16 = relu(conv(A15, H16));
	FeatureMap A17 = relu(conv(A16, H17));
	FeatureMap A18 = pool_max(A17);

	// -----------
	// Section 6:
	// -----------
	cout << "\nSection 6: layers 19-23" << R[18] << "x" << C[18] << "x" << D[18] << " -> " << R[19] << "x" << C[19] << "x" << D[19] << " -> "
		<< " -> " << R[20] << "x" << C[20] << "x" << D[20] << " -> " << R[21] << "x" << C[21] << "x" << D[21]
		<< " -> " << R[22] << "x" << C[22] << "x" << D[22] << " -> " << R[23] << "x" << C[23] << "x" << D[23] << "\n";
	cout << "From Darknet: 13x13x512 -> 13x13x1024 -> 13x13x512 -> 13x13x1024 -> 13x13x512 -> 13x13x1024 \n";
	FeatureMap A19 = relu(conv(A18, H19));
	FeatureMap A20 = relu(conv(A19, H20));
	FeatureMap A21 = relu(conv(A20, H21));
	FeatureMap A22 = relu(conv(A21, H22));
	FeatureMap A23 = relu(conv(A22, H23));

	// End CPU Timing
	QueryPerformanceCounter(&end_CPU);
	milliseconds_CPU = (end_CPU.QuadPart - start_CPU.QuadPart) *
		1000.0 / frequency_CPU.QuadPart;
	seconds_CPU = milliseconds_CPU / 1000;
	minutes_CPU = seconds_CPU / 60;
	fprintf(stderr, "\nCPU Time = %.3f milliseconds", milliseconds_CPU);
	fprintf(stderr, "\nCPU Time = %.3f seconds", seconds_CPU);
	fprintf(stderr, "\nCPU Time = %.3f minutes\n\n", minutes_CPU);
	*/
	cout << "\n\nCompleted CNN\n\n";
	getchar();


    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
