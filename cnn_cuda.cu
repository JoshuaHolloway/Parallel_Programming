#include "hip/hip_runtime.h"
#include "header.h"
#include <vector>
using std::vector;
//===============
struct FeatureMap
{
	float* val = nullptr;
	size_t rows;
	size_t cols;
	size_t channels;
	size_t num_elems;

	FeatureMap(size_t rows, size_t cols, size_t channels)
	{
		num_elems = rows * cols * channels;
		this->rows = rows;
		this->cols = cols;
		this->channels = channels;
		val = new float[num_elems];
	}


	// Easier indexing is achieved with (channel, row, col)
	void set(size_t i, size_t j, size_t k, float val)
	{
		this->val[i * rows * cols + j * cols + k] = val;
	}

	float at(size_t i, size_t j, size_t k) //(channel, row, col)
	{
		return val[i * rows * cols + j * cols + k];
	}

	void ones()
	{
		for (int i = 0; i < channels; ++i)
		{
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					this->val[i * rows * cols + j * cols + k] = 1;
				}
			}
		}
	}

	void zeros()
	{
		for (int i = 0; i < channels; ++i)
		{
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					this->val[i * rows * cols + j * cols + k] = 0;
				}
			}
		}
	}

	void count()
	{
		for (int i = 0; i < channels; ++i)
		{
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					this->val[i * rows * cols + j * cols + k] = i * rows * cols + j * cols + k;
				}
			}
		}
	}

	void print()
	{
		for (int i = 0; i < channels; ++i)
		{
			cout << "\n ----------------- \n";
			cout << "Slice " << i << " \n";
			for (int j = 0; j < rows; ++j)
			{
				for (int k = 0; k < cols; ++k)
				{
					cout << val[i * rows * cols + j * cols + k] << " ";
				}
				cout << "\n";
			}
		}
	}
};
//---------------
struct Tensor
{

	size_t dim1;
	size_t dim2;
	size_t dim3;
	size_t dim4;

	// pixel data
	float* val = nullptr;

	// (filters, channels, rows, cols)
	size_t filters;		// dim 1
	size_t channels;	// dim 2
	size_t rows;			// dim 3
	size_t cols;			// dim 4

										// total number of pixels
	size_t num_elems;


	// Vector of feature maps - tensor = set of feature maps:
	std::vector<FeatureMap> tensor;

	// Constructor and destructor
	Tensor(size_t filters, size_t channels, size_t rows, size_t cols)
	{
		num_elems = rows * cols * channels * filters;

		val = new float[num_elems];

		this->filters = filters;			this->dim1 = filters;
		this->channels = channels;		this->dim2 = channels;
		this->rows = rows;						this->dim3 = rows;
		this->cols = cols;						this->dim4 = cols;
	}
	~Tensor()
	{
		delete[] val;
	}

	// Easier indexing is achieved with (output_channel, input_channel, row, col)
	void set(size_t i, size_t j, size_t k, size_t l, float val)
	{
		// i: dim1 - filters
		// j: dim2 - channels
		// k: dim3 - rows
		// l: dim4 - cols

		// 3D linear index: i * rows * cols + j * cols + k
		// i * cols * rows + j * cols + k
		// (i * dim3 * dim2) + (j * dim3) + k

		// 4D linear index:
		// (i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l

		this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] = val;
	}

	float at(size_t i, size_t j, size_t k, size_t l) //(filters, channel, row, col)
	{
		return val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l];
	}

	void ones()
	{
		for (int i = 0; i < dim1; ++i)
		{
			for (int j = 0; j < dim2; ++j)
			{
				for (int k = 0; k < dim3; ++k)
				{
					for (int l = 0; l < dim4; ++l)
						this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] = 1;
				}
			}
		}
	}

	void zeros()
	{
		for (int i = 0; i < dim1; ++i)
		{
			for (int j = 0; j < dim2; ++j)
			{
				for (int k = 0; k < dim3; ++k)
				{
					for (int l = 0; l < dim4; ++l)
						this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] = 0;
				}
			}
		}
	}

	void count()
	{
		for (int i = 0; i < dim1; ++i)
		{
			for (int j = 0; j < dim2; ++j)
			{
				for (int k = 0; k < dim3; ++k)
				{
					for (int l = 0; l < dim4; ++l)
					{
						this->val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l]
							= (i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l;

					}
				}
			}
		}
	};

	void print()
	{
		for (int i = 0; i < filters; ++i)
		{
			cout << "\n ----------------- \n";
			cout << "Volume " << i << " \n";
			for (int j = 0; j < channels; ++j)
			{
				cout << "\n ----------------- \n";
				cout << "Slice " << i << " \n";
				for (int k = 0; k < rows; ++k)
				{
					for (int l = 0; l < cols; ++l)
					{
						cout << val[(i * dim4 * dim3 * dim2) + (j * dim4 * dim3) + (k * dim4) + l] << " ";
					}
					cout << "\n";
				}
			}
		}
	}
};
//-------------------------------------
FeatureMap conv(FeatureMap x, Tensor h)
{
	// 'same' 2D conv with 3D feature maps with implicit matrix slice addition
	// Zero-padding is also implicit
	//
	// Input: One 3D feature map and one 4D tensor (set of filters)
	// Output: One 3D feature map
	FeatureMap y(x.rows, x.cols, h.filters);
	for (int idq = 0; idq < h.filters; ++idq) // out_channels
	{
		for (int idy = 0; idy < x.rows; ++idy) // out_rows
		{
			for (int idx = 0; idx < x.cols; ++idx) // out_cols
			{
				float Pvalue = 0.0f;
				for (int idz = 0; idz < x.channels; ++idz) // input_channels
				{

					int M_start_point = idy - h.rows / 2;
					int N_start_point = idx - h.cols / 2;
					for (int i = 0; i < h.rows; ++i) // filter_rows
					{
						for (int j = 0; j < h.cols; ++j) // filter_cols
						{
							if ((M_start_point + i >= 0 && M_start_point + i < x.rows)
								&& (N_start_point + j >= 0 && N_start_point + j < x.cols))
							{
								Pvalue += x.at(idz, M_start_point + i, N_start_point + j) * h.at(idq, idz, i, j);
							}
						}
					}
					y.set(idq, idy, idx, Pvalue);
				}
			}
		}
	}
	return y;
}
//-------------------------------
FeatureMap pool_ave(FeatureMap x)
{
	const size_t H = x.rows;
	const size_t W = x.cols;
	const size_t M = x.channels;
	const size_t K = 2; // downsampling factor

	FeatureMap S(H / K, W / K, x.channels); // rows, cols, channels

	for (int m = 0; m < M; ++m)  // channels
	{
		for (int h = 0; h < H / K; ++h) // rows
		{
			for (int w = 0; w < W / K; ++w)
			{
				float temp = 0.0f;
				for (int p = 0; p < K; ++p)
				{
					for (int q = 0; q < K; ++q)
					{
						temp += x.at(m, K*h + p, K*w + q) / float(K*K);
					}
				}
				S.set(m, h, w, temp);
			}
		}
	}

	return S;
}
//-------------------------------
FeatureMap pool_max(FeatureMap x)
{
	// downsampling factor:
	const size_t K = 2;

	// output downsampled feature map:
	FeatureMap y(x.rows / K, x.cols / K, x.channels); // rows, cols, channels

	for (int i = 0; i < x.channels; ++i)
	{
		for (int j = 0; j < x.rows; j += K)
		{
			for (int k = 0; k < x.cols; k += K)
			{
				// Search inside the KxK block for max value
				int max = 0;
				for (int jj = j; jj < j + K; ++jj)
				{
					for (int kk = k; kk < k + K; ++kk)
					{
						if (jj == j && kk == k)
						{
							max = x.at(i, jj, kk);
						}
						else
						{
							if (x.at(i, jj, kk) > max)
								max = x.at(i, jj, kk);
						} // end if-else
					}// end for over kk
				} // end for over jj
					//y[i][j / 2][k / 2] = max;
				y.set(i, j / 2, k / 2, max);
			} // end for over k
		} // end for over j
	} // end for over i
	return y;
}
//---------------------------
FeatureMap relu(FeatureMap z)
{
	for (int i = 0; i < z.channels; ++i)
	{
		for (int j = 0; j < z.rows; ++j)
		{
			for (int k = 0; k < z.cols; ++k)
			{
				// Leaky ReLu
				if (z.at(i, j, k) < 0)
					z.set(i, j, k, 0.1*z.at(i, j, k));
			}
		}
	}
	return z;
}
//=============================================================================
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
//-----------------------------------------------------------------------------
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
//--------
//--------------------------------------
void save_image(const char* output_filename,
	float* buffer,
	int height,
	int width) {
	cv::Mat output_image(height, width, CV_32FC3, buffer);
	// Make negative values zero.
	cv::threshold(output_image,
		output_image,
		/*threshold=*/0,
		/*maxval=*/0,
		cv::THRESH_TOZERO);
	cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
	output_image.convertTo(output_image, CV_8UC3);
	cv::imwrite(output_filename, output_image);
}
//--------------------------------------
class Layer
{
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

public:
	hipdnnTensorDescriptor_t input_descriptor;
	hipdnnTensorDescriptor_t output_descriptor;
	hipdnnFilterDescriptor_t kernel_descriptor;
	hipdnnConvolutionDescriptor_t convolution_descriptor;
	hipdnnConvolutionFwdAlgo_t convolution_algorithm;

};
//--------------------------------------
class Network
{
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

private:
	cv::Mat image;
	hipdnnHandle_t cudnn;

	vector<Layer> layers;

	//vector<hipdnnTensorDescriptor_t> input_descriptor_vect;
	//vector<hipdnnTensorDescriptor_t> output_descriptor_vect;
	//vector<hipdnnFilterDescriptor_t> kernel_descriptor_vect;
	//vector<hipdnnConvolutionDescriptor_t> convolution_descriptor_vect;
	//vector<hipdnnConvolutionFwdAlgo_t> convolution_algorithm_vect;
		 
public:
	Network(const cv::Mat& image)
	{
		this->image = image;

		/// cuDNN Step 1: Create context object

		// Create context object

		checkCUDNN(hipdnnCreate(&cudnn));
	}
	~Network()
	{
	}

	void do_stuff()
	{
		/// cuDNN Step 2: Create objects to store input tensor, output tensor, and output tensor

		// Input tensor - 1
		layers.push_back(Layer{});

		// input_descriptor_vect.push_back(hipdnnTensorDescriptor_t{}); // place new one in vector
		//checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor_vect[0]));
		checkCUDNN(hipdnnCreateTensorDescriptor(&layers[0].input_descriptor));

		checkCUDNN(hipdnnSetTensor4dDescriptor(layers[0].input_descriptor, //input_descriptor_1,
			/*format=*/HIPDNN_TENSOR_NHWC,
			/*dataType=*/HIPDNN_DATA_FLOAT,
			/*batch_size=*/1,
			/*channels=*/3,
			/*image_height=*/image.rows,
			/*image_width=*/image.cols));

		// Output tensor - 1
		//output_descriptor_vect.push_back(hipdnnTensorDescriptor_t{}); // place new one in vector
		
		checkCUDNN(hipdnnCreateTensorDescriptor(&layers[0].output_descriptor));//output_descriptor_vect[0]));// output_descriptor_1));
		checkCUDNN(hipdnnSetTensor4dDescriptor(layers[0].output_descriptor, // output_descriptor_vect[0],//output_descriptor_1,
			/*format=*/HIPDNN_TENSOR_NHWC,
			/*dataType=*/HIPDNN_DATA_FLOAT,
			/*batch_size=*/1,
			/*channels=*/3,
			/*image_height=*/image.rows,
			/*image_width=*/image.cols));

		// Input tensor - 2
		//hipdnnTensorDescriptor_t input_descriptor_2;
		//input_descriptor_vect.push_back(hipdnnTensorDescriptor_t{}); // place new one in vector
		layers.push_back(Layer{});

		checkCUDNN(hipdnnCreateTensorDescriptor(&layers[1].input_descriptor));//input_descriptor_vect[1]));
		checkCUDNN(hipdnnSetTensor4dDescriptor(layers[1].input_descriptor, //input_descriptor_vect[1],
			/*format=*/HIPDNN_TENSOR_NHWC,
			/*dataType=*/HIPDNN_DATA_FLOAT,
			/*batch_size=*/1,
			/*channels=*/3,
			/*image_height=*/image.rows,
			/*image_width=*/image.cols));

		// Copy output of 1 descriptor into input of 2 descriptor:
		//input_descriptor_vect[1] = input_descriptor_vect[0];
		layers[1].input_descriptor = layers[0].input_descriptor;

		// Filter tensor
		//hipdnnFilterDescriptor_t kernel_descriptor_1;
		//kernel_descriptor_vect.push_back(hipdnnFilterDescriptor_t{});


		checkCUDNN(hipdnnCreateFilterDescriptor(&layers[0].kernel_descriptor));//kernel_descriptor_vect[0]));//kernel_descriptor_1));
		checkCUDNN(hipdnnSetFilter4dDescriptor(layers[0].kernel_descriptor,//kernel_descriptor_vect[0],
			/*dataType=*/HIPDNN_DATA_FLOAT,
			/*format=*/HIPDNN_TENSOR_NCHW,
			/*out_channels=*/3,
			/*in_channels=*/3,
			/*kernel_height=*/3,
			/*kernel_width=*/3));

		// Describe the conv kernel
		//hipdnnConvolutionDescriptor_t convolution_descriptor_1;
		//convolution_descriptor_vect.push_back(hipdnnConvolutionDescriptor_t{});
		checkCUDNN(hipdnnCreateConvolutionDescriptor(&layers[0].convolution_descriptor));//convolution_descriptor_vect[0]));//convolution_descriptor_1));
		checkCUDNN(hipdnnSetConvolution2dDescriptor(layers[0].convolution_descriptor, //convolution_descriptor_vect[0],//convolution_descriptor_1,
			/*pad_height=*/1,
			/*pad_width=*/1,
			/*vertical_stride=*/1,
			/*horizontal_stride=*/1,
			/*dilation_height=*/1,
			/*dilation_width=*/1,
			/*mode=*/HIPDNN_CROSS_CORRELATION,
			/*computeType=*/HIPDNN_DATA_FLOAT));

		// More detailed description of the convolution algorithm we want to use:
		//hipdnnConvolutionFwdAlgo_t convolution_algorithm_1;
		//convolution_algorithm_vect.push_back(hipdnnConvolutionFwdAlgo_t{});
		checkCUDNN(
			hipdnnGetConvolutionForwardAlgorithm(cudnn,
				layers[0].input_descriptor,//input_descriptor_vect[0], //input_descriptor_1,
				layers[0].kernel_descriptor,//kernel_descriptor_vect[0],//kernel_descriptor_1,			
				layers[0].convolution_descriptor,//convolution_descriptor_vect[0], //convolution_descriptor_1,
				layers[0].output_descriptor, //output_descriptor_vect[0], //output_descriptor_1,
				HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
				/*memoryLimitInBytes=*/0,
				&layers[0].convolution_algorithm));//convolution_algorithm_vect[0]));// convolution_algorithm_1));

		// Physical memory to operate on
		size_t workspace_bytes = 0;
		checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
			layers[0].input_descriptor,//input_descriptor_vect[0], //input_descriptor_1,
			layers[0].kernel_descriptor,//kernel_descriptor_vect[0],// kernel_descriptor_1,
			layers[0].convolution_descriptor,//convolution_descriptor_vect[0], //convolution_descriptor_1,
			layers[0].output_descriptor,//output_descriptor_vect[0], //output_descriptor_1,
			layers[0].convolution_algorithm,//convolution_algorithm_vect[0], // convolution_algorithm_1,
			&workspace_bytes));
		std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
			<< std::endl;

		// Dimensions from OpenCV input image
		int batch_size = 1;
		int channels = image.channels();
		int height = image.rows;
		int width = image.cols;
		cout << "\n batch_size = " << batch_size << " channels = " << channels << " rows = " << height << " cols = " << width << "\n";

		// Dimensions from cuDNN for confirmation
		hipdnnGetConvolution2dForwardOutputDim(
			layers[0].convolution_descriptor,//convolution_descriptor_vect[0], //convolution_descriptor_1,
			layers[0].input_descriptor, //input_descriptor_vect[0], //input_descriptor_1,
			layers[0].kernel_descriptor,//kernel_descriptor_vect[0],//kernel_descriptor_1,
			&batch_size,
			&channels,
			&height,
			&width);
		cout << "\n batch_size = " << batch_size << " channels = " << channels << " rows = " << height << " cols = " << width << "\n";


		// Allocate device memory
		void* d_workspace{ nullptr };
		hipMalloc(&d_workspace, workspace_bytes);

		int image_bytes = batch_size * channels * height * width * sizeof(float);

		float* d_input_1{ nullptr };
		hipMalloc(&d_input_1, image_bytes);
		hipMemcpy(d_input_1, image.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);

		float* d_input_2{ nullptr };
		hipMalloc(&d_input_2, image_bytes);
		hipMemcpy(d_input_2, image.ptr<float>(0), image_bytes, hipMemcpyHostToDevice);

		
		
		float* d_output_1{ nullptr };
		hipMalloc(&d_output_1, image_bytes);
		hipMemset(d_output_1, 0, image_bytes);



		float* d_output_2{ nullptr };
		hipMalloc(&d_output_2, image_bytes);
		hipMemset(d_output_2, 0, image_bytes);




		// Mystery kernel
		const float kernel_template[3][3] = {
			{ 1,  1, 1 },
			{ 1, -8, 1 },
			{ 1,  1, 1 }
		};

		float h_kernel[3][3][3][3];
		for (int kernel = 0; kernel < 3; ++kernel) {
			for (int channel = 0; channel < 3; ++channel) {
				for (int row = 0; row < 3; ++row) {
					for (int column = 0; column < 3; ++column) {
						h_kernel[kernel][channel][row][column] = kernel_template[row][column];
					}
				}
			}
		}

		float* d_kernel_1{ nullptr };
		hipMalloc(&d_kernel_1, sizeof(h_kernel));
		hipMemcpy(d_kernel_1, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

		// Do conv
		const float alpha = 1, beta = 0;
		checkCUDNN(hipdnnConvolutionForward(cudnn,
			&alpha,
			layers[0].input_descriptor,//input_descriptor_vect[0], //input_descriptor_1,
			d_input_1,
			layers[0].kernel_descriptor, //kernel_descriptor_vect[0],//kernel_descriptor_1,
			d_kernel_1,
			layers[0].convolution_descriptor, //convolution_descriptor_vect[0],//convolution_descriptor_1,
			layers[0].convolution_algorithm, //convolution_algorithm_vect[0], //convolution_algorithm_1,
			d_workspace,
			workspace_bytes,
			&beta,
			layers[0].output_descriptor, //output_descriptor_vect[0], //output_descriptor_1,
			d_output_1));


		// Copy output of first layer into input of second layer
		hipMemcpy(d_input_2, d_output_1, image_bytes, hipMemcpyDeviceToDevice);


		// Copy DEVICE -> HOST
		float* h_output = new float[image_bytes];
		hipMemcpy(h_output, d_output_1, image_bytes, hipMemcpyDeviceToHost);

		// Do something with h_output ...
		save_image("convolved_img.jpg",
			h_output,
			height,
			width);

		cv::imshow("After first conv", cv::imread("convolved_img.jpg"));


		
		checkCUDNN(hipdnnConvolutionForward(cudnn,
			&alpha,
			layers[1].input_descriptor,//input_descriptor_vect[1], //input_descriptor_2,
			d_input_2,
			layers[0].kernel_descriptor, //kernel_descriptor_vect[0], //kernel_descriptor_1, // change to 2
			d_kernel_1,
			layers[0].convolution_descriptor, //convolution_descriptor_vect[0], //convolution_descriptor_1,
			layers[0].convolution_algorithm,// convolution_algorithm_vect[0], //convolution_algorithm_1,
			d_workspace,
			workspace_bytes,
			&beta,
			layers[0].output_descriptor, //output_descriptor_vect[0], //output_descriptor_1, // change to 2
			d_output_2));




		// Copy DEVICE -> HOST
		//float* h_output = new float[image_bytes];
		hipMemcpy(h_output, d_output_2, image_bytes, hipMemcpyDeviceToHost);

		// Do something with h_output ...
		save_image("convolved_img.jpg",
			h_output,
			height,
			width);

		cv::imshow("After second conv", cv::imread("convolved_img.jpg"));
		cv::waitKey(0);

		// Free memory
		delete[] h_output;
		hipFree(d_kernel_1);
		hipFree(d_input_1);
		hipFree(d_output_1);
		hipFree(d_workspace);

		//hipdnnDestroyTensorDescriptor(input_descriptor_vect[0]); //input_descriptor_1);
		//hipdnnDestroyTensorDescriptor(input_descriptor_vect[1]); //input_descriptor_1);
		//hipdnnDestroyTensorDescriptor(output_descriptor_1);
		//hipdnnDestroyFilterDescriptor(kernel_descriptor_1);
		//hipdnnDestroyConvolutionDescriptor(convolution_descriptor_1);

		hipdnnDestroy(cudnn);
	}
};
//--------------------------------------
int kernel_wrapper(const cv::Mat& image)
{

	Network net(image);
	net.do_stuff();

	return 0;
}
